#include <iostream>

#include "Window.hpp"
#include "Camera.hpp"
#include "Sphere.hpp"
#include "Plane.hpp"
#include "Scene.hpp"
#include "Kernel.hpp"

#define RETURN_SUCCESS 0
#define RETURN_FAILURE 1

#define WINDOW_WIDTH 1024
#define WINDOW_HEIGHT 768

int
main(int ac, char** av)
{
    (void)ac;
    (void)av;

    // Create window
    Window      window(WINDOW_WIDTH, WINDOW_HEIGHT);
    if (!window.valid())
    {
        return RETURN_FAILURE;
    }

    // Create camera
    Camera      camera;
    camera.setPosition(0.0, 0.0, 15.0);
    camera.setPlane(WINDOW_WIDTH, WINDOW_HEIGHT);
    camera.setCache();

    // Create Scene
    Scene       scene;
    // Walls
    Plane*      backWall = new Plane(0.0, 0.0, -20.0, 0.0, 0.0, 1.0);
    backWall->setDiffuse(0.5);
    backWall->setSpecular(0.4);
    // backWall->setReflectance(0.5);
    Plane*      floorWall = new Plane(0.0, -5.0, 0.0, 0.0, 1.0, 0.0, 50, 50, 50);
    floorWall->setDiffuse(0.9);
    floorWall->setSpecular(0.1);
    floorWall->setReflectance(0.05);
    Plane*      ceilWall = new Plane(0.0, 10.0, 0.0, 0.0, -1.0, 0.0);
    Plane*      leftWall = new Plane(-10.0, 0.0, 0.0, 1.0, 0.0, 0.0, 200, 75, 75);
    Plane*      rightWall = new Plane(10.0, 0.0, 0.0, -1.0, 0.0, 0.0, 100, 100, 200);
    // Spheres
    // Reflective sphere
    Sphere*     leftSphere = new Sphere(-5.0, -2.0, -15.0, 3.0);
    leftSphere->setDiffuse(0.0);
    leftSphere->setSpecular(0.4);
    leftSphere->setReflectance(1.0);
    // Small sphere
    Sphere*     smallSphere1 = new Sphere(1.0, -2.5, -15.0, 0.25, 255, 75, 75);
    smallSphere1->setDiffuse(1.0);
    smallSphere1->setSpecular(0.4);
    // Small sphere
    Sphere*     smallSphere2 = new Sphere(2.0, -2.5, -15.0, 0.25, 255, 75, 75);
    smallSphere2->setDiffuse(1.0);
    smallSphere2->setSpecular(0.4);
    // Small sphere
    Sphere*     smallSphere3 = new Sphere(3.0, -2.5, -15.0, 0.25, 255, 75, 75);
    smallSphere3->setDiffuse(1.0);
    smallSphere3->setSpecular(0.4);
    // Small sphere
    Sphere*     smallSphere4 = new Sphere(4.0, -2.5, -15.0, 0.25, 255, 75, 75);
    smallSphere4->setDiffuse(1.0);
    smallSphere4->setSpecular(0.4);
    // Small sphere
    Sphere*     smallSphere5 = new Sphere(5.0, -2.5, -15.0, 0.25, 255, 75, 75);
    smallSphere5->setDiffuse(1.0);
    smallSphere5->setSpecular(0.4);
    // Transmittive sphere
    Sphere*     rightSphere = new Sphere(3.0, -2.0, -8.0, 3.0, 200, 200, 200);
    rightSphere->setDiffuse(0.2);
    rightSphere->setSpecular(0.0);
    rightSphere->setTransmittance(0.8);
    rightSphere->setRefractive(1.4);
    // Lights
    Light*      ceilLight = new Light(0.0, 8.0, -15.0);
    Light*      backLight = new Light(0.0, 8.0, 10.0);
    // Add items to scene
    scene.addObject(backWall);
    scene.addObject(floorWall);
    scene.addObject(ceilWall);
    scene.addObject(leftWall);
    scene.addObject(rightWall);
    scene.addObject(leftSphere);
    scene.addObject(rightSphere);
    scene.addObject(smallSphere1);
    scene.addObject(smallSphere2);
    scene.addObject(smallSphere3);
    scene.addObject(smallSphere4);
    scene.addObject(smallSphere5);
    scene.addLight(ceilLight);
    scene.addLight(backLight);

    // Cuda device variables initialization
    Camera*                 d_camera;
    uint32_t*               d_pixels;
    hipMalloc(&d_camera, sizeof(Camera));
    hipMalloc(&d_pixels, WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uint32_t));
    hipMemcpy(d_pixels, window.pixels(), WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uint32_t), hipMemcpyHostToDevice);

    // Copy scene from host to device
    Scene*                  d_scene;
    Object**                d_objects;
    Light**                 d_lights;
    hipMalloc(&d_scene, sizeof(Scene));
    hipMalloc(&d_objects, MAX_NB_OBJECTS * sizeof(Object*));
    hipMalloc(&d_lights, MAX_NB_LIGHTS * sizeof(Light*));

    Object**                objects;
    Light**                 lights;
    objects = new Object*[MAX_NB_OBJECTS];
    lights = new Light*[MAX_NB_LIGHTS];
    // Copy objects
    for (uint64_t i = 0; i < scene.objectsSize(); i++)
    {
        if (scene.objects()[i]->type() == ObjectType::Sphere)
        {
            hipMalloc(&objects[i], sizeof(Sphere));
            hipMemcpy(objects[i], scene.objects()[i], sizeof(Sphere), hipMemcpyHostToDevice);
        }
        else if (scene.objects()[i]->type() == ObjectType::Plane)
        {
            hipMalloc(&objects[i], sizeof(Plane));
            hipMemcpy(objects[i], scene.objects()[i], sizeof(Plane), hipMemcpyHostToDevice);
        }
    }
    hipMemcpy(d_objects, objects, MAX_NB_OBJECTS * sizeof(Object*), hipMemcpyHostToDevice);
    // Copy lights
    for (uint64_t i = 0; i < scene.lightsSize(); i++)
    {
        hipMalloc(&lights[i], sizeof(Light));
        hipMemcpy(lights[i], scene.lights()[i], sizeof(Light), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_lights, lights, MAX_NB_LIGHTS * sizeof(Light*), hipMemcpyHostToDevice);
    for (uint64_t i = 0; i < scene.objectsSize(); i++)
    {
        delete scene.objects()[i];
    }
    for (uint64_t i = 0; i < scene.lightsSize(); i++)
    {
        delete scene.lights()[i];
    }
    delete[] scene.objects();
    delete[] scene.lights();
    scene.setObjects(d_objects);
    scene.setLights(d_lights);
    hipMemcpy(d_scene, &scene, sizeof(scene), hipMemcpyHostToDevice);

    // Main loop
    while (!window.over())
    {
        // Handle updates
        window.pollEvents();

        // Upate camera position
        window.update(camera);
        hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

        // Rendering kernel
        dim3 threadsPerBlock(16, 16);
        dim3 grid(WINDOW_WIDTH / threadsPerBlock.x, WINDOW_HEIGHT / threadsPerBlock.y);
        renderKernel<<<grid, threadsPerBlock>>>(d_camera, d_scene, d_pixels);

        // Copy pixels from device to renderer
        hipMemcpy(window.pixels(), d_pixels, WINDOW_WIDTH * WINDOW_HEIGHT * sizeof(uint32_t), hipMemcpyDeviceToHost);
        window.render();
    }

    // Host clean
    for (uint64_t i = 0; i < scene.objectsSize(); i++)
    {
        hipFree(objects[i]);
    }
    for (uint64_t i = 0; i < scene.lightsSize(); i++)
    {
        hipFree(lights[i]);
    }
    delete[] objects;
    delete[] lights;

    // Device clean
    hipFree(d_pixels);
    hipFree(d_camera);
    hipFree(d_scene);
    hipFree(d_objects);
    hipFree(d_lights);

    return RETURN_SUCCESS;
}
